#include "hip/hip_runtime.h"
#include "include/CUDASearch.h"
#include "include/CUDAHelper.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

// CUDA facts:
//
// On devices of compute capability 2.x and beyond, 32-bit integer multiplication is natively supported,
// but 24-bit integer multiplication is not. __[u]mul24 is therefore implemented using multiple instructions
// and should not be used.
//
// Integer division and modulo operation are costly: below 20 instructions on devices of compute capability 2.x and
// higher. They can be replaced with bitwise operations in some cases: If n is a power of 2, (i/n) is equivalent to
// (i>>log2(n)) and (i%n) is equivalent to (i&(n-1)); the compiler will perform these conversions if n is literal.

namespace clusteringgraph {
	/// CUDA kernel for simple byte-per-cell world evaluation.
	///
	/// @param lifeData  Linearized 2D array of life data with byte-per-cell density.
	/// @param worldWidth  Width of life world in cells (bytes).
	/// @param worldHeight  Height of life world in cells (bytes).
	/// @param resultLifeData  Result buffer in the same format as input.
	__global__ void simpleSearchKernel(const float* weightArray, const int* destArray, const int* numArray,
			const int* offsetArray, const ulong* clusterArray, const float* funcArray, uint n, uint m,
		ulong* destClusterArray, float* destPosImbArray, float* destNegImbArray, ulong nc) {
		

		// compute new objective function value
		// test
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		
	int i = idx % 50;
	float negativeSum = 0.0, positiveSum = 0.0;
	ulong count = offsetArray[i] + numArray[i];
	for (ulong edgenum = offsetArray[i]; edgenum < count; edgenum++) {
		int targ = destArray[edgenum];
		float weight = weightArray[edgenum];
		if(clusterArray[targ] == clusterArray[i]) {  // same cluster
			if(weight < 0) {
				negativeSum += (-1) * weight;
			}
		} else {
			if(weight > 0) {
				positiveSum += weight;
			}
		}
	}
	destPosImbArray[idx] += positiveSum;
	destNegImbArray[idx] += negativeSum;

	// iterates over in-edges of vertex i -- TODO implement this
	/*
	DirectedGraph::in_edge_iterator in_i, in_end;
	// std::cout << "in-edges of " << i << ": ";
	for (tie(in_i, in_end) = in_edges(i, g.graph); in_i != in_end; ++in_i) {
		e = *in_i;
		Vertex src = source(e, g.graph), targ = target(e, g.graph);
		double weight = ((Edge*)in_i->get_property())->weight;
		if(cluster[src.id]) {
			if(weight < 0) {
				negativeSum += fabs(weight);
			}
		} else {
			if(weight > 0) {
				positiveSum += weight;
			}
		}
	} */
		
		
		// for each vertex i, tries to move i to another cluster in myNeighborClusterList[i]
		// For each node i in cluster(k1)
		/*
		for (unsigned long i = randomUtil.next(initialSearchIndex, finalSearchIndex), cont2 = 0; cont2 < numberOfVerticesInInterval; cont2++) {
			// vertex i is in cluster(k1)
			ulong k1 = clusterArray[i];
			// Option 1: node i is moved from k1 to another existing cluster k2 != k1
			for (ulong k2 = 0; k2 < nc; k2++) {  // cluster(k2)
				if(k2 != k1) {
					// removes node i from cluster1 and inserts in cluster2
					
				}
			}
			// Option 2: node i is moved to a new cluster, alone
			cTemp.removeNodeFromCluster(*g, problem, i, k1);
			// clusterArray[i] = nc++;
			
		} */
		
		/*
		uint worldSize = worldWidth * worldHeight;
		
		for (uint cellId = blockIdx.x * blockDim.x + threadIdx.x;
				cellId < worldSize;
				cellId += blockDim.x * gridDim.x) {

			uint x = cellId % worldWidth;
			uint yAbs = cellId - x;
			
			// Count alive cells.
			uint aliveCells = lifeData[xLeft + yAbsUp] + lifeData[x + yAbsUp] + lifeData[xRight + yAbsUp]
				+ lifeData[xLeft + yAbs] + lifeData[xRight + yAbs]
				+ lifeData[xLeft + yAbsDown] + lifeData[x + yAbsDown] + lifeData[xRight + yAbsDown];

			resultLifeData[x + yAbs] = aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]) ? 1 : 0;
		} */
	}
	
	/// Runs a kernel for simple byte-per-cell world evaluation.
	bool runSimpleSearchKernel(thrust::host_vector<float>& h_weights, thrust::host_vector<int>& h_dest,
			thrust::host_vector<int>& h_numedges, thrust::host_vector<int>& h_offset,
			thrust::host_vector<unsigned long>& h_mycluster, thrust::host_vector<float>& h_functionValue,
			ulong n, ulong m,
			thrust::host_vector<unsigned long>& h_destcluster, thrust::host_vector<float>& h_destPosFunctionValue,
			thrust::host_vector<float>& h_destNegFunctionValue, ushort threadsCount, ulong nc) {

		thrust::device_vector<float> d_weights = h_weights;  // edge weights
		thrust::device_vector<int> d_dest = h_dest;  // edge destination (vertex j)
		thrust::device_vector<int> d_numedges = h_numedges;  // number of edges of each vertex i
		thrust::device_vector<int> d_offset = h_offset;  // initial edge number for vertex i
		thrust::device_vector<float> d_functionValue = h_functionValue;
		thrust::device_vector<unsigned long> d_mycluster = h_mycluster;
		// destination vectors
		thrust::device_vector<float> d_destPosFunctionValue = h_destPosFunctionValue;
		thrust::device_vector<float> d_destNegFunctionValue = h_destNegFunctionValue;
		thrust::device_vector<unsigned long> d_destCluster = h_destcluster;
	
		float* weightArray = thrust::raw_pointer_cast( &d_weights[0] );
		int* destArray = thrust::raw_pointer_cast( &d_dest[0] );
		int* numArray = thrust::raw_pointer_cast( &d_numedges[0] );
		int* offsetArray = thrust::raw_pointer_cast( &d_offset[0] );
		unsigned long* clusterArray = thrust::raw_pointer_cast( &d_mycluster[0] );
		float* funcArray = thrust::raw_pointer_cast( &d_functionValue[0] );
		unsigned long* destClusterArray = thrust::raw_pointer_cast( &d_destCluster[0] );
		float* destPosImbArray = thrust::raw_pointer_cast( &d_destPosFunctionValue[0] );
		float* destNegImbArray = thrust::raw_pointer_cast( &d_destNegFunctionValue[0] );

		size_t reqBlocksCount = (n * (nc - 1)) / threadsCount;
		ushort blocksCount = (ushort)std::min((size_t)32768, reqBlocksCount);
		// <<<blocksCount, threadsCount>>>
		simpleSearchKernel<<<1, threadsCount>>>(weightArray, destArray, numArray, offsetArray, 
				clusterArray, funcArray, uint(n), uint(m), destClusterArray, destPosImbArray, destNegImbArray, nc);
		
		checkCudaErrors(hipDeviceSynchronize());

		h_destcluster = d_destCluster;
		h_destPosFunctionValue = d_destPosFunctionValue;
		h_destNegFunctionValue = d_destNegFunctionValue;
		
		return true;
	}
		
}
